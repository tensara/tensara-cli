#include <hip/hip_runtime.h>

__global__ void reference_vector_add(const float* input1, const float* input2, float* output, size_t n) {
    const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        output[idx] = input1[idx] + input2[idx];
    }

}

extern "C" void solution(const float* d_input1, const float* d_input2, float* d_output, size_t n) {
    const int block_size = 256;
    const int num_blocks = (n + block_size - 1) / block_size;
    reference_vector_add<<<num_blocks, block_size>>>(d_input1, d_input2, d_output, n);
} 