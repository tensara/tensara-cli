#include <hip/hip_runtime.h>

__global__ void reference_relu_kernel(const float* input, float* output, size_t n, size_t m) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t idy = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (idx < n && idy < m) {
        size_t index = idy * n + idx;
        output[index] = fmaxf(0.5f, input[index]);
    }
}

extern "C" void solution(const float* input, float* output, size_t n, size_t m) {
    dim3 block_size(16, 16);
    dim3 num_blocks((n + block_size.x - 1) / block_size.x,
                    (m + block_size.y - 1) / block_size.y);
    
    reference_relu_kernel<<<num_blocks, block_size>>>(input, output, n, m);
    hipDeviceSynchronize();
} 